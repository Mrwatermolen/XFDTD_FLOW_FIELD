#include "hip/hip_runtime.h"
#include <xfdtd/boundary/pml.h>
#include <xfdtd/monitor/field_monitor.h>
#include <xfdtd/monitor/movie_monitor.h>
#include <xfdtd/nffft/nffft_frequency_domain.h>
#include <xfdtd/simulation/simulation.h>
#include <xfdtd/waveform_source/tfsf_3d.h>

#include <filesystem>
#include <xfdtd_cuda/simulation/simulation_hd.cuh>
#include <xtensor/xnpy.hpp>

#include "argparse.hpp"
#include "flow_field.h"

int main(int argc, char** argv) {
  auto start_time = std::chrono::high_resolution_clock::now();

  xfdtd::MpiSupport::setMpiParallelDim(1, 2, 2);
  xfdtd::MpiSupport::instance(argc, argv);
  constexpr auto data_path_str = "./tmp/data/flow_field_cuda";
  const auto data_path = std::filesystem::path{data_path_str};

  auto program = argparse::ArgumentParser("flow_field_cuda");
  program.add_argument("-f_p", "--flow_field_path")
      .help("flow field data path")
      .required();
  program.add_argument("-g", "--cuda_grid_dim")
      .help("cuda grid dim")
      .default_value(std::vector<unsigned int>{128, 128, 2})
      .nargs(3)
      .action([](const std::string& value) { return std::stoi(value); });
  program.add_argument("-b", "--cuda_block_dim")
      .help("cuda block dim")
      .default_value(std::vector<unsigned int>{2, 2, 64})
      .nargs(3)
      .action([](const std::string& value) { return std::stoi(value); });
  try {
    program.parse_args(argc, argv);
  } catch (const std::runtime_error& err) {
    std::cout << err.what() << std::endl;
    std::cout << program;
    exit(0);
  }

  auto flow_field = std::make_shared<xfdtd::FlowField>(
      "flow_field", program.get<std::string>("--flow_field_path"));
  auto vector_to_dim = [](const auto& vec) {
    dim3 dim;
    dim.x = vec[0];
    dim.y = vec[1];
    dim.z = vec[2];
    return dim;
  };
  auto grid_dim =
      vector_to_dim(program.get<std::vector<int>>("--cuda_grid_dim"));
  auto block_dim =
      vector_to_dim(program.get<std::vector<int>>("--cuda_block_dim"));

  auto&& shape = flow_field->flowFieldShape();
  auto&& cube = shape.wrappedCube();

  constexpr xfdtd::Real dl{20e-3};

  auto domain_shape = xfdtd::Cube{
      xfdtd::Vector{-10 * dl + cube->originX(), -10 * dl + cube->originY(),
                    -10 * dl + cube->originZ()},
      xfdtd::Vector{20 * dl + cube->sizeX(), 20 * dl + cube->sizeY(),
                    20 * dl + cube->sizeZ()}};

  auto domain = std::make_shared<xfdtd::Object>(
      "domain", std::make_unique<xfdtd::Cube>(domain_shape),
      xfdtd::Material::createAir());

  auto s = xfdtd::Simulation{dl, dl, dl, 0.9, xfdtd::ThreadConfig{1, 1, 1}};
  s.addObject(domain);
  s.addObject(flow_field);

  constexpr auto l_min = dl * 20;
  constexpr auto f_max = 3e8 / l_min;
  constexpr auto tau = l_min / 6e8;
  constexpr auto t_0 = 4.5 * tau;
  constexpr xfdtd::Index tfsf_start = 13;
  auto tfsf{std::make_shared<xfdtd::TFSF3D>(
      tfsf_start, tfsf_start, tfsf_start, xfdtd::constant::PI / 2, 0, 1,
      xfdtd::Waveform::gaussian(tau, t_0))};

  s.addWaveformSource(tfsf);

  auto movie_ez_xy{std::make_shared<xfdtd::MovieMonitor>(
      std::make_unique<xfdtd::FieldMonitor>(
          std::make_unique<xfdtd::Cube>(
              xfdtd::Vector{domain_shape.originX(), domain_shape.originY(), 0},
              xfdtd::Vector{domain_shape.sizeX(), domain_shape.sizeY(), dl}),
          xfdtd::EMF::Field::EZ, "", ""),
      20, "movie_ez_xy", (data_path / "movie_ez_xy").string())};

  s.addMonitor(movie_ez_xy);

  s.addBoundary(std::make_shared<xfdtd::PML>(8, xfdtd::Axis::Direction::XN));
  s.addBoundary(std::make_shared<xfdtd::PML>(8, xfdtd::Axis::Direction::XP));
  s.addBoundary(std::make_shared<xfdtd::PML>(8, xfdtd::Axis::Direction::YN));
  s.addBoundary(std::make_shared<xfdtd::PML>(8, xfdtd::Axis::Direction::YP));
  s.addBoundary(std::make_shared<xfdtd::PML>(8, xfdtd::Axis::Direction::ZN));
  s.addBoundary(std::make_shared<xfdtd::PML>(8, xfdtd::Axis::Direction::ZP));

  auto nf2ff_fd = std::make_shared<xfdtd::NFFFTFrequencyDomain>(
      11, 11, 11, xfdtd::Array<xfdtd::Real>{0.8 * f_max});
  s.addNF2FF(nf2ff_fd);

  auto s_hd = xfdtd::cuda::SimulationHD{&s};
  s_hd.setGridDim(grid_dim);
  s_hd.setBlockDim(block_dim);
  s_hd.run(1000);

  nf2ff_fd->setOutputDir((data_path / "fd").string());
  nf2ff_fd->processFarField(
      xfdtd::constant::PI * 0.5,
      xt::linspace<double>(-xfdtd::constant::PI, xfdtd::constant::PI, 360),
      "xy");

  auto time = tfsf->waveform()->time();
  auto incident_wave_data = tfsf->waveform()->value();
  if (!xfdtd::MpiSupport::instance().isRoot()) {
    return 0;
  }
  xt::dump_npy((data_path / "time.npy").string(), time);
  xt::dump_npy((data_path / "incident_wave.npy").string(), incident_wave_data);

  auto end_time = std::chrono::high_resolution_clock::now();

  auto duration_ms = std::chrono::duration_cast<std::chrono::milliseconds>(
                         end_time - start_time)
                         .count();

  std::stringstream ss;
  ss << "Elapsed time: " << duration_ms << " ms" << " " << duration_ms / 1000.0
     << " s" << " " << duration_ms / 60000.0 << " min";

  std::cout << ss.str() << "\n";

  return 0;
}