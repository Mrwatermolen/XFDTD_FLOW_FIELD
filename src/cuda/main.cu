#include "hip/hip_runtime.h"
#include <xfdtd/boundary/pml.h>
#include <xfdtd/monitor/field_monitor.h>
#include <xfdtd/monitor/movie_monitor.h>
#include <xfdtd/nffft/nffft_frequency_domain.h>
#include <xfdtd/simulation/simulation.h>
#include <xfdtd/waveform_source/tfsf_3d.h>

#include <ase_reader/ase_reader.hpp>
#include <filesystem>
#include <xfdtd_cuda/simulation/simulation_hd.cuh>
#include <xfdtd_model/grid_model.hpp>
#include <xfdtd_model/model_object.hpp>
#include <xfdtd_model/model_shape.hpp>
#include <xtensor/xnpy.hpp>

#include "argparse.hpp"
#include "flow_field.h"

int main(int argc, char** argv) {
  auto start_time = std::chrono::high_resolution_clock::now();

  xfdtd::MpiSupport::setMpiParallelDim(1, 2, 2);
  xfdtd::MpiSupport::instance(argc, argv);
  constexpr auto data_path_str = "./tmp/data/flow_field_cuda";
  const auto data_path = std::filesystem::path{data_path_str};

  auto program = argparse::ArgumentParser("flow_field_cuda");
  program.add_argument("-ase").help("ASE file path").required();
  program.add_argument("-f_p", "--flow_field_path")
      .help("flow field data path")
      .required();
  program.add_argument("-r", "--resolution")
      .help("resolution")
      .default_value(20e-3)
      .action([](const std::string& value) { return std::stod(value); });
  program.add_argument("-g", "--cuda_grid_dim")
      .help("cuda grid dim")
      .default_value(std::vector<unsigned int>{128, 128, 2})
      .nargs(3)
      .action([](const std::string& value) { return std::stoi(value); });
  program.add_argument("-b", "--cuda_block_dim")
      .help("cuda block dim")
      .default_value(std::vector<unsigned int>{2, 2, 64})
      .nargs(3)
      .action([](const std::string& value) { return std::stoi(value); });
  try {
    program.parse_args(argc, argv);
  } catch (const std::runtime_error& err) {
    std::cout << err.what() << std::endl;
    std::cout << program;
    exit(0);
  }

  const auto dl = program.get<xfdtd::Real>("--resolution");

  std::cout << "Resolution: " << dl << "\n";

  auto metal_info_ss = std::stringstream{};
  auto metal_vertices_ss = std::stringstream{};
  auto metal_elements_ss = std::stringstream{};

  {
    auto ase_path_str = program.get<std::string>("-ase");
    auto ase_path = std::filesystem::path{ase_path_str};
    if (!std::filesystem::exists(ase_path)) {
      std::cerr << "ASE file not found: " << ase_path_str << std::endl;
      exit(1);
    }

    auto ase_reader = ase_reader::ASEReader{};
    ase_reader.read(ase_path.string());
    ase_reader.setPrecision(8);

    {
      constexpr auto unit = xfdtd::unit::Length::Millimeter;
      const auto delta_l = xfdtd::model::ModelShape<unit>::standardToUnit(dl);
      for (const auto& o : ase_reader.objects()) {
        auto info_ss = std::stringstream{};
        auto vertices_ss = std::stringstream{};
        auto elements_ss = std::stringstream{};
        auto grid_model = xfdtd::model::GridModel{};
        o.write(info_ss, vertices_ss, elements_ss);
        grid_model.read(info_ss, vertices_ss, elements_ss);

        std::cout << "Object: " << o.name() << "\n";
        std::cout << "Region: " << "\n";
        std::cout << "  Origin: " << grid_model.triangularModelInfo().minX()
                  << " " << grid_model.triangularModelInfo().minY() << " "
                  << grid_model.triangularModelInfo().minZ() << "\n";
        std::cout << "  Size: " << grid_model.triangularModelInfo().sizeX()
                  << " " << grid_model.triangularModelInfo().sizeY() << " "
                  << grid_model.triangularModelInfo().sizeZ() << "\n";
        std::cout << " End: " << grid_model.triangularModelInfo().maxX() << " "
                  << grid_model.triangularModelInfo().maxY() << " "
                  << grid_model.triangularModelInfo().maxZ() << "\n";

        if (o.name() == "metal") {
          metal_info_ss << info_ss.str();
          metal_vertices_ss << vertices_ss.str();
          metal_elements_ss << elements_ss.str();
        }
      }

      auto info_ss = std::stringstream{};
      auto vertices_ss = std::stringstream{};
      auto elements_ss = std::stringstream{};
      ase_reader.write(info_ss, vertices_ss, elements_ss);
      auto grid_model = xfdtd::model::GridModel{};
      grid_model.read(info_ss, vertices_ss, elements_ss);
    }
  }

  auto model_shape = std::make_unique<
      xfdtd::model::ModelShape<xfdtd::unit::Length::Millimeter>>(
      metal_info_ss, metal_vertices_ss, metal_elements_ss);
  std::cout << "Model shape Wrapping: "
            << model_shape->wrappedCube()->toString() << "\n";

  auto model_object = std::make_shared<xfdtd::model::ModelObject>(
      "metal", std::move(model_shape), xfdtd::Material::createPec());

  auto flow_field = std::make_shared<xfdtd::FlowField>(
      "flow_field", program.get<std::string>("--flow_field_path"));
  auto vector_to_dim = [](const auto& vec) {
    dim3 dim;
    dim.x = vec[0];
    dim.y = vec[1];
    dim.z = vec[2];
    return dim;
  };
  auto grid_dim =
      vector_to_dim(program.get<std::vector<int>>("--cuda_grid_dim"));
  auto block_dim =
      vector_to_dim(program.get<std::vector<int>>("--cuda_block_dim"));

  auto&& shape = flow_field->flowFieldShape();
  auto&& cube = shape.wrappedCube();

  auto domain_shape = xfdtd::Cube{
      xfdtd::Vector{-10 * dl + cube->originX(), -10 * dl + cube->originY(),
                    -10 * dl + cube->originZ()},
      xfdtd::Vector{20 * dl + cube->sizeX(), 20 * dl + cube->sizeY(),
                    20 * dl + cube->sizeZ()}};

  auto domain = std::make_shared<xfdtd::Object>(
      "domain", std::make_unique<xfdtd::Cube>(domain_shape),
      xfdtd::Material::createAir());

  auto s = xfdtd::Simulation{dl, dl, dl, 0.9, xfdtd::ThreadConfig{1, 1, 1}};
  s.addObject(domain);
  s.addObject(flow_field);
  s.addObject(model_object);

  constexpr auto l_min = 20e-3 * 20;
  constexpr auto f_max = 3e8 / l_min;
  constexpr auto tau = l_min / 6e8;
  constexpr auto t_0 = 4.5 * tau;
  constexpr xfdtd::Index tfsf_start = 13;
  auto tfsf{std::make_shared<xfdtd::TFSF3D>(
      tfsf_start, tfsf_start, tfsf_start, xfdtd::constant::PI / 2, 0, 1,
      xfdtd::Waveform::gaussian(tau, t_0))};

  s.addWaveformSource(tfsf);

  auto movie_ez_xy{std::make_shared<xfdtd::MovieMonitor>(
      std::make_unique<xfdtd::FieldMonitor>(
          std::make_unique<xfdtd::Cube>(
              xfdtd::Vector{domain_shape.originX(), domain_shape.originY(), 0},
              xfdtd::Vector{domain_shape.sizeX(), domain_shape.sizeY(), dl}),
          xfdtd::EMF::Field::EZ, "", ""),
      20, "movie_ez_xy", (data_path / "movie_ez_xy").string())};

  s.addMonitor(movie_ez_xy);

  s.addBoundary(std::make_shared<xfdtd::PML>(8, xfdtd::Axis::Direction::XN));
  s.addBoundary(std::make_shared<xfdtd::PML>(8, xfdtd::Axis::Direction::XP));
  s.addBoundary(std::make_shared<xfdtd::PML>(8, xfdtd::Axis::Direction::YN));
  s.addBoundary(std::make_shared<xfdtd::PML>(8, xfdtd::Axis::Direction::YP));
  s.addBoundary(std::make_shared<xfdtd::PML>(8, xfdtd::Axis::Direction::ZN));
  s.addBoundary(std::make_shared<xfdtd::PML>(8, xfdtd::Axis::Direction::ZP));

  auto nf2ff_fd = std::make_shared<xfdtd::NFFFTFrequencyDomain>(
      11, 11, 11, xfdtd::Array<xfdtd::Real>{0.8 * f_max});
  s.addNF2FF(nf2ff_fd);

  auto s_hd = xfdtd::cuda::SimulationHD{&s};
  s_hd.setGridDim(grid_dim);
  s_hd.setBlockDim(block_dim);
  s_hd.run(1000);

  nf2ff_fd->setOutputDir((data_path / "fd").string());
  nf2ff_fd->processFarField(
      xfdtd::constant::PI * 0.5,
      xt::linspace<double>(-xfdtd::constant::PI, xfdtd::constant::PI, 360),
      "xy");

  auto time = tfsf->waveform()->time();
  auto incident_wave_data = tfsf->waveform()->value();
  if (!xfdtd::MpiSupport::instance().isRoot()) {
    return 0;
  }
  xt::dump_npy((data_path / "time.npy").string(), time);
  xt::dump_npy((data_path / "incident_wave.npy").string(), incident_wave_data);

  auto end_time = std::chrono::high_resolution_clock::now();

  auto duration_ms = std::chrono::duration_cast<std::chrono::milliseconds>(
                         end_time - start_time)
                         .count();

  std::stringstream ss;
  ss << "Elapsed time: " << duration_ms << " ms" << " " << duration_ms / 1000.0
     << " s" << " " << duration_ms / 60000.0 << " min";

  std::cout << ss.str() << "\n";

  return 0;
}